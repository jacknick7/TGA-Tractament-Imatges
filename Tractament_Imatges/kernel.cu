#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define SIZE 32

#ifndef PINNED
#define PINNED 0
#endif


// Matriz por Matriz
// C(NxM) <- A(NxP) * B (PxM)

__global__ void Kernel01(int N, int M, int P, float *A, float *B, float *C) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < N && col < M) {
		float tmp = 0.0;
		for (int k = 0; k < P; k++)
			tmp += A[row*P + k] * B[k*M + col];
		C[row*M + col] = tmp;
	}
}



void InitM(int N, int M, float *Mat);
int TestMM(int N, int M, int P, float *A, float *B, float *C);


// Invocacion:
// ./ejecutable N M P test
// TAM es el la dimension de las matrices
// test == 'Y', comprueba que el resultado sea correcto
// test == 'N', NO comprueba que el resultado (Util para tomar tiempos)
// Por defecto, N = 639, M = 641, P = 1023, test == 'N'

int main(int argc, char** argv)
{
	unsigned int N, M, P;
	unsigned int numBytesC, numBytesA, numBytesB;
	unsigned int nBlocksN, nBlocksM, nThreads;

	float TiempoTotal, TiempoKernel;
	hipEvent_t E0, E1, E2, E3;

	float *h_A, *h_B, *h_C;
	float *d_A, *d_B, *d_C;

	char test;

	// Dimension de las matrices NxM, NxP, PxM y comprobacion resultado
	
	N = 64;
	M = 1024;
	P = 128;
	test = 'Y';
	//else { printf("Usage: ./exe N M P test\n"); exit(0); }

	// numero de Threads en cada dimension 
	nThreads = SIZE;

	// numero de Blocks en cada dimension 
	nBlocksN = (N + nThreads - 1) / nThreads;
	nBlocksM = (M + nThreads - 1) / nThreads;

	numBytesC = N * M * sizeof(float);
	numBytesA = N * P * sizeof(float);
	numBytesB = P * M * sizeof(float);

	dim3 dimGrid(nBlocksM, nBlocksN, 1);
	dim3 dimBlock(nThreads, nThreads, 1);

	hipEventCreate(&E0);
	hipEventCreate(&E1);
	hipEventCreate(&E2);
	hipEventCreate(&E3);

	if (PINNED) {
		// Obtiene Memoria [pinned] en el host
		hipHostMalloc((float**)&h_A, numBytesA);
		hipHostMalloc((float**)&h_B, numBytesB);
		hipHostMalloc((float**)&h_C, numBytesC);
	}
	else {
		// Obtener Memoria en el host
		h_A = (float*)malloc(numBytesA);
		h_B = (float*)malloc(numBytesB);
		h_C = (float*)malloc(numBytesC);
	}

	// Inicializa las matrices
	InitM(N, P, h_A);
	InitM(P, M, h_B);

	hipEventRecord(E0, 0);
	hipEventSynchronize(E0);

	// Obtener Memoria en el device
	hipMalloc((float**)&d_A, numBytesA);
	hipMalloc((float**)&d_B, numBytesB);
	hipMalloc((float**)&d_C, numBytesC);

	// Copiar datos desde el host en el device 
	hipMemcpy(d_A, h_A, numBytesA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, numBytesB, hipMemcpyHostToDevice);

	hipEventRecord(E1, 0);
	hipEventSynchronize(E1);

	// Ejecutar el kernel 
	Kernel01 << <dimGrid, dimBlock >> > (N, M, P, d_A, d_B, d_C);

	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);

	// Obtener el resultado desde el host 
	hipMemcpy(h_C, d_C, numBytesC, hipMemcpyDeviceToHost);

	// Liberar Memoria del device 
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	hipEventRecord(E3, 0);
	hipEventSynchronize(E3);

	hipEventElapsedTime(&TiempoTotal, E0, E3);
	hipEventElapsedTime(&TiempoKernel, E1, E2);
	printf("\nKERNEL 01\n");
	printf("Dimensiones: %dx%d <- %dx%d * %dx%d\n", N, M, N, P, P, M);
	printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
	printf("nBlocks: %dx%d (%d)\n", nBlocksM, nBlocksN, nBlocksN*nBlocksM);
	if (PINNED) printf("Usando Pinned Memory\n");
	else printf("NO usa Pinned Memory\n");
	printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
	printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
	printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float)N * (float)M * (float)P) / (1000000.0 * TiempoTotal));
	printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float)N * (float)M * (float)P) / (1000000.0 * TiempoKernel));

	hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

	if (test == 'N')
		printf("NO TEST\n");
	else  if (TestMM(N, M, P, h_A, h_B, h_C))
		printf("TEST PASS\n");
	else
		printf("TEST FAIL\n");

	if (PINNED) {
		hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);
	}
	else {
		free(h_A); free(h_B); free(h_C);
	}

}


void InitM(int N, int M, float *Mat) {
	int i;
	for (i = 0; i < N*M; i++)
		Mat[i] = rand() / (float)RAND_MAX;

}

int error(float a, float b) {
	float tmp;

	tmp = abs(a - b) / abs(min(a, b));

	if (tmp > 0.0001) return 1;
	else  return 0;

}

int TestMM(int N, int M, int P, float *A, float *B, float *C) {
	int i, j, k;
	float tmp;
	for (i = 0; i < N; i++)
		for (j = 0; j < M; j++) {
			tmp = 0.0;
			for (k = 0; k < P; k++)
				tmp = tmp + A[i*P + k] * B[k*M + j];
			if (error(tmp, C[i*M + j])) {
				printf("%d:%d: %f - %f = %f \n", i, j, tmp, C[i*M + j], abs(tmp - C[i*M + j]));
				return 0;
			}
		}

	return 1;
}

